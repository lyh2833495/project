#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ void test(int *thI,int *blI,int *blD,int *tidd)
{
	int tid = threadIdx.x+blockIdx.x *blockDim.x;
	tidd[tid] = tid;
	thI[tid] = threadIdx.x;
	blI[tid] = blockIdx.x;
	blD[tid] = blockDim.x;
}

int main(void)
{
	int *dev_thI, *dev_blI,*dev_blD,int *dev_tidd;
	int thI[4],blI[4],blD[4],tidd[4];

	hipMalloc((void**)&dev_thI, 4 * sizeof(int));
	hipMalloc((void**)&dev_blI, 4 * sizeof(int));
	hipMalloc((void**)&dev_blD, 4 * sizeof(int));
	hipMalloc((void**)&dev_tidd, 4 * sizeof(int));
	for (int i = 0; i < 4; i++)
	{
		thI[i] = 999;
		blI[i] = 999;
		blD[i] = 999;
		tidd[i] = 999;
	}
	//hipMemcpy(dev_a, a, 4 * sizeof(int), hipMemcpyHostToDevice);
	test<< <2, 2>> >(dev_thI,dev_blI,dev_blD,dev_tidd);
	hipMemcpy(thI, dev_thI, 4 * sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(blI, dev_blI, 4 * sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(blD, dev_blD, 4 * sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(tidd, dev_tidd, 4 * sizeof(int), hipMemcpyDeviceToHost);
	for (int i = 0; i < 4; i++)
	{
		printf("This Tid=%d\n", tidd[i]);
		printf("This ThreadIdx.x=%d \n", thI[i]);
		printf("This BlockIdx.x=%d \n", blI[i]);
		printf("This BlockDim.x=%d \n\n", blD[i]);
	}
	hipFree(dev_thI);
	hipFree(dev_blI);
	hipFree(dev_blD);
	hipFree(dev_tidd);
	return 0;
}


