#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include<time.h>
#include <stdio.h>

#define N 4
#define NN 2000
__global__ void mat(int *a, int *b, int *c)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

int main(void)

{
	int a[NN][NN], b[NN][NN];
	long long c[NN][NN];
	int *dev_a, *dev_b;
	long long *dev_c;
	clock_t start, end;
	int i, j, t, k,n=1000, ptime;
	hipMalloc((void**)&dev_a, NN*sizeof(int));
	hipMalloc((void**)&dev_b, NN* sizeof(int));
	hipMalloc((void**)&dev_c, NN* sizeof(long));

	for(i=0;i<n;i++)
	{ 
		t = i + 1;
		for (j = 0; j < n; j++)
		{
			a[i][j] = t++;
			b[i][j] = 1;

		}
	}
	
	hipMemcpy(dev_a, a, NN* sizeof(int*), hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b, NN*sizeof(int*), hipMemcpyHostToDevice);
	hipMemcpy(dev_c, c, NN*sizeof(long*), hipMemcpyHostToDevice);

	start = clock();
	mat << <1, 10 >> > (dev_a, dev_b, dev_c);
	end = clock();
	ptime = end - start;
	printf("Time=%d", ptime);

	
	
}
