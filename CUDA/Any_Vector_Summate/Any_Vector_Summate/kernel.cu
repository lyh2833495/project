#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include"D:\CUDA_files\cuda_by_example\common\book.h"

//#include <stdio.h>


#define N (34*1024)
__global__ void add(int *a, int *b, int *c)
{
	int tid = threadIdx.x+blockIdx.x*blockDim.x;
	while(tid < N)
	{
		c[tid] = a[tid] + b[tid];
		tid += blockDim.x*gridDim.x;
		
	}
}
int main(void)
{
	int a[N], b[N], c[N];  
	int *dev_a, *dev_b, *dev_c;

	//Distribure memroy on GPU
	hipMalloc((void**)&dev_a, N*sizeof(int));
	hipMalloc((void**)&dev_b, N*sizeof(int));
	hipMalloc((void**)&dev_c, N*sizeof(int));

	//Set array a&b on CPU
	for (int i = 0; i < N; i++)
	{
		a[i] = i;
		b[i] = i*i;
	}

	//Copy array a&b to GPU
	hipMemcpy(dev_a, a, N*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b, N*sizeof(int), hipMemcpyHostToDevice);

	//Using kernel
	////////////////////128-bolck 128-thread /////////////////
	add << <128,128>> >(dev_a, dev_b, dev_c);

	//Copy array c to CPU
	hipMemcpy(c, dev_c, N*sizeof(int), hipMemcpyDeviceToHost);

	//Show the result
	for (int i = 0; i < N; i++)
	{
		printf("%d+%d=%d\n", a[i], b[i], c[i]);
	}

	//Confirm GPU complete the work
	/*bool success = true;
	for (int i = 0; i < N; i++)
	{
		if (a[i] + b[i] != c[i])
		{
			printf("ERROR:%d+%d!=%d\n", a[i], b[i], c[i]);
			success = false;
		}
	}
	if (success){ printf("We Did It!\n"); }*/

	//Free the memory on GPU
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);
	
	return 0;
}