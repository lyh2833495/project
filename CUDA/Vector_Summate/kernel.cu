#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include"D:\CUDA_files\cuda_by_example\common\book.h"

#include <stdio.h>


#define N 10
__global__ void add(int *a, int *b, int *c)
{
	int tid = threadIdx.x;
	if (tid < N)
	{
		c[tid] = a[tid] + b[tid];

	}

}
int main(void)
{
	int a[N], b[N], c[N];
	int *dev_a, *dev_b, *dev_c;
	
	//Distribure memroy on GPU
	hipMalloc((void**)&dev_a, N*sizeof(int));
	hipMalloc((void**)&dev_b, N*sizeof(int));
	hipMalloc((void**)&dev_c, N*sizeof(int));

	//Set array a&b on CPU
	for (int i = 0; i < N; i++)
	{
		a[i] = i;
		b[i] = i*i;
	}

	//Copy array a&b to GPU
	hipMemcpy(dev_a, a, N*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b, N*sizeof(int), hipMemcpyHostToDevice);

	//Using kernel
	////////////////////1-bolck N-thread /////////////////
	add << <1, N >> >(dev_a, dev_b, dev_c);

	//Copy array c to CPU
	hipMemcpy(c, dev_c, N*sizeof(int), hipMemcpyDeviceToHost);

	//Show the result
	for (int i = 0; i < N; i++)
	{
		printf("%d+%d=%d\n", a[i], b[i], c[i]);
	}
	
	//Free the memory on GPU
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);

	return 0;
}