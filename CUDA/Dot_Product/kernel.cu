#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include"D:\CUDA_files\cuda_by_example\common\book.h"
#include "D:\CUDA_files\cuda_by_example\appendix_a\lock.h"
#include <stdio.h>

#define imin(a,b) (a<b?a:b)

const int N = 33 * 1024;
const int threadsPerBlock = 256;
const int blocksPerGrid = imin(32, (N + threadsPerBlock - 1) / threadsPerBlock);

__global__ void dot(float *a, float *b, float *c)
{
	__shared__ float cache[threadsPerBlock];
	int tid = threadIdx.x + blockIdx.x*blockDim.x;
	int cacheIndex = threadIdx.x;//CacheID=threadID
	float temp = 0;////////
	while (tid < N){
		temp += a[tid] * b[tid];
		tid += blockDim.x*gridDim.x;
	}

	cache[cacheIndex] = temp;//Set value in cache
	__syncthreads();//Wait every threads finish work
	int i = blockDim.x / 2;//For this caculate threadPerBlock must /2=0
	while (i != 0){
		if (cacheIndex < i)
			cache[cacheIndex] += cache[cacheIndex + i];
		__syncthreads();
		i /= 2;
	}
	if (cacheIndex == 0){
		c[blockIdx.x] = cache[0];
	}
}
int main(void)
{
	float *a, *b, c, *partial_c;
	float *dev_a, *dev_b, *dev_partial_c;

	//Set memory on CPU
	a = (float*)malloc(N*sizeof(float));
	b = (float*)malloc(N*sizeof(float));
	partial_c = (float*)malloc(blocksPerGrid*sizeof(float));

	//Set memory on GPU
	hipMalloc((void**)&dev_a, N*sizeof(float));
	hipMalloc((void**)&dev_b, N*sizeof(float));
	hipMalloc((void**)&dev_partial_c, N*sizeof(float));

	//Fill Host's memory(set array a&b value)
	for (int i = 0; i < N; i++)
	{
		a[i] = i;
		b[i] = i * 2;
	}

	//Copy array a&b to GPU
	hipMemcpy(dev_a, a, N*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b, N*sizeof(float), hipMemcpyHostToDevice);

	dot << <blocksPerGrid, threadsPerBlock >> >(dev_a, dev_b, dev_partial_c);

	//Copy array c to CPU
	hipMemcpy(partial_c, dev_partial_c, blocksPerGrid*sizeof(float), hipMemcpyDeviceToHost);

	//Finish final caculate at CPU
	c = 0;
	for (int i = 0; i < blocksPerGrid; i++){
		c += partial_c[i];
	}
    #define sum_squares(x) (x*(x+1)*(2*x+1)/6)
	printf("Does GPU value %.6g=%.6g?\n", c, 2 * sum_squares((float)(N - 1)));

	//Free memory on GPU
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_partial_c);

	//Free memory on CPU
	free(a);
	free(b);
	free(partial_c);




}